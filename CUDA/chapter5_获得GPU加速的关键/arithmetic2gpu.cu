#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cmath>
#include <cstdio>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 10;
const real x0 = 100.0;
__global__ void arithmetic(real *d_x, const real x0, const int N);

int main(int argc, char *argv[])
{
    if (argc != 2)
    {
        printf("usage: %s N\n", argv[0]);
    }
    const int N = atoi(argv[1]);
    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    const int M = sizeof(real) * N;
    real *h_x = (real *)malloc(M);
    real *d_x;

    CHECK(hipMalloc((void **)&d_x, M));

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        for (int n = 0; n < N; ++n)
        {
            h_x[n] = 0.0;
        }
        CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        arithmetic<<<grid_size, block_size>>>(d_x, x0, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    free(h_x);
    CHECK(hipFree(d_x));

    return 0;
}

__global__ void arithmetic(real *d_x, const real x0, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        real x_tmp = d_x[n];
        while (sqrt(x_tmp) < x0)
        {
            ++x_tmp;
        }
        d_x[n] = x_tmp;
    }
}