#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h>
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
    const real EPSILON = 1.0e-15;
#else
    typedef float real;
    const real EPSILON = 1.0e-6f;
#endif

const int NUM_REPEATS = 10;
const real a = 1.23;
const real b = 2.34;
const real c = 3.57;

void __global__ add(const real *x, const real *y, real *z, const int N);
void check(const real *z, const int N);

int main(int argc, char *argv[])
{
    const int N = 100000000;
    const int M = sizeof(real) * N;
    real *h_x = (real*)malloc(M);
    real *h_y = (real*)malloc(M);
    real *h_z = (real*)malloc(M);

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    real *d_x, *d_y, *d_z;
    CHECK(hipMalloc((void **)&d_x, M));
    CHECK(hipMalloc((void **)&d_y, M));
    CHECK(hipMalloc((void **)&d_z, M));

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

        add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

        CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));

    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));
    return 0;
}

void __global__ add(const real *x, const real *y, real *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
}

void check(const real *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

// nvprof ./add3memcpy