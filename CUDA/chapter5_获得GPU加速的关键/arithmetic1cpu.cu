#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cmath>
#include <cstdio>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 10;
const real x0 = 100.0;
void arithmetic(real *x, const real x0, const int N);

int main(int argc, char *argv[])
{
    const int N = 10000;
    const int M = sizeof(real) * N;
    real *x = (real*)malloc(M);

    float t_sum = 0;
    float t2_sum = 0;

    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        for (int n = 0; n < N; ++n)
        {
            x[n] = 0.0;
        }

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        arithmetic(x, x0, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    free(x);
    return 0;
}

void arithmetic(real *x, const real x0, const int N)
{
    for (int n = 0; n < N; ++n)
    {
        real x_tmp = x[n];
        while (sqrt(x_tmp) < x0)
        {
            ++x_tmp;
        }
        x[n] = x_tmp;
    }
}