#include "error.cuh"
#include <cstdio>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 20;
void timing(const real *x, const int N);
real reduce(const real *x, const int N);

int main(int argc, char *argv[])
{
    const int N = 100000000;
    const int M = sizeof(real) * N;
    real *x = (real *)malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.23;
    }

    timing(x, N);

    free(x);
}

void timing(const real *x, const int N)
{
    real sum = 0;

    for (int repeat = 0; repeat < NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        sum = reduce(x, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    printf("sum = %f.\n", sum);
}

real reduce(const real *x, const int N)
{
    real sum = 0.0;
    for (int n = 0; n < N; ++n)
    {
        sum += x[n];
    }
    return sum;
}