#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cstdio>

__device__ int d_x = 1;
__device__ int d_y[2];

void __global__ my_kernel(void)
{
    d_y[0] += d_x;
    d_y[1] += d_x;
    printf("d_x = %d, d_y[0] = %d, d_y[1] = %d.\n", d_x, d_y[0], d_y[1]);
}

int main(int argc, char *argv[])
{
    int h_y[2] = {10, 20};
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_y), h_y, sizeof(int) * 2));

    my_kernel<<<1, 1>>>();
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpyFromSymbol(h_y, HIP_SYMBOL(d_y), sizeof(int) * 2));
    printf("h_y[0] = %d, h_y[1] = %d.\n", h_y[0], h_y[1]);

    return 0;
}

