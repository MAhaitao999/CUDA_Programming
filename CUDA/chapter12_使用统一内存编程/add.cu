#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cmath>
#include <cstdio>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;
void __global__ add(const double *x, const double *y, double *z);
void check(const double *z, const int N);

int main(int argc, char *argv[])
{
    const int N = 100000000;
    const int M = sizeof(double) * N;
    double *x, *y, *z;
    CHECK(hipMallocManaged((void **)&x, M));
    CHECK(hipMallocManaged((void **)&y, M));
    CHECK(hipMallocManaged((void **)&z, M));

    for (int n = 0; n < N; ++n)
    {
        x[n] = a;
        y[n] = b;
    }

    const int block_size = 128;
    const int grid_size = N / block_size;
    add<<<grid_size, block_size>>>(x, y, z);

    CHECK(hipDeviceSynchronize());
    check(z, N);

    CHECK(hipFree(x));
    CHECK(hipFree(y));
    CHECK(hipFree(z));
    return 0;
}

void __global__ add(const double *x, const double *y, double *z)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    z[n] = x[n] + y[n];
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}