#include "error.cuh"
#include <cstdio>
#include <cstdint>

const int N = 30;

int main(int argc, char *argv[])
{
    for (int n = 1; n <= N; ++n)
    {
        const size_t size = size_t(n) * 1024 * 1024 * 1024;
        uint16_t *x;
#ifdef UNIFIED
    CHECK(hipMallocManaged(&x, size));
    CHECK(hipFree(x));
    printf("Allocated %d GB unified memory without touch.\n", n);
#else
    CHECK(hipMalloc(&x, size));
    CHECK(hipFree(x));
    printf("Allocate %d GB device memory.\n", n);
#endif
    }

    return 0;
}

