
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    printf("Hello world from block %d and thread %d\n", bid, tid);
}

int main(int argc, char *argv[])
{
    hello_from_gpu<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}