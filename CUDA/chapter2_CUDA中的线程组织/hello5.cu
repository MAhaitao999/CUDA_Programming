
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_from_gpu()
{
    const int b = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    printf("Hello World from block-%d and thread-(%d, %d)!\n", b, tx, ty);
}

int main(int argc, char *argv[])
{
    const dim3 block_size(2, 4);
    hello_from_gpu<<<1, block_size>>>();

    hipDeviceSynchronize();

    return 0;
}