
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_from_gpu()
{
    printf("Hello World from the GPU!\n");
}

int main(int argc, char *argv[])
{
    hello_from_gpu<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}