#include "hip/hip_runtime.h"
#include "common.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <omp.h>

/*
 * An example of using OpenMP to parallelize the creation of CUDA work in
 * multiple streams. This example using n_streams OpenMP threads to launch 4
 * kernels in each stream. Note the new pragma introduced, #pragma omp parallel.
 */

#define N 300000000
#define NSTREAM 4

__global__ void kernel_1()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_2()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_3()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_4()
{
    double sum = 0.0;

    for(int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

int main(int argc, char **argv)
{
    int n_streams = NSTREAM;
    int isize = 1;
    int iblock = 1;
    int bigcase = 0;

    // get argument from command line
    if (argc > 1) n_streams = atoi(argv[1]);

    if (argc > 2) bigcase = atoi(argv[2]);

    float elapsed_time;

    // set up max connectioin
    char iname[] = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv (iname, "4", 1);
    char *ivalue =  getenv (iname);
    printf ("%s = %s\n", iname, ivalue);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams=%d\n", dev, deviceProp.name,
           n_streams);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
    {
        if (deviceProp.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                    "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // Allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(n_streams * sizeof(
                                hipStream_t));

    for (int i = 0 ; i < n_streams ; i++)
    {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    // run kernel with more threads
    if (bigcase == 1)
    {
        iblock = 512;
        isize = 1 << 12;
    }

    // set up execution configuration
    dim3 block (iblock);
    dim3 grid  (isize / iblock);
    printf("> grid %d block %d\n", grid.x, block.x);

    // creat events
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // record start event
    CHECK(hipEventRecord(start, 0));

    // dispatch job with depth first ordering using OpenMP
    omp_set_num_threads(n_streams);
    #pragma omp parallel
    {
        int i = omp_get_thread_num();
        kernel_1<<<grid, block, 0, streams[i]>>>();
        kernel_2<<<grid, block, 0, streams[i]>>>();
        kernel_3<<<grid, block, 0, streams[i]>>>();
        kernel_4<<<grid, block, 0, streams[i]>>>();
    }

    // record stop event
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %.3fs\n",
           elapsed_time / 1000.0f);

    // release all stream
    for (int i = 0 ; i < n_streams ; i++)
    {
        CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    // destroy events
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    // reset device
    CHECK(hipDeviceReset());

    return 0;
}