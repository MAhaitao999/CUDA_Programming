#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cmath>
#include <cstdio>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 10;
const int N = 1 << 22;
const int M = sizeof(real) * N;
const int MAX_NUM_STREAMS = 64;
hipStream_t streams[MAX_NUM_STREAMS];

void timing(const real *h_x, const real *h_y, real *h_z,
            real *d_x, real *d_y, real *d_z,
            const int num
           );

int main(int argc, char *argv[])
{
    real *h_x, *h_y, *h_z;
    CHECK(hipHostMalloc(&h_x, M));
    CHECK(hipHostMalloc(&h_y, M));
    CHECK(hipHostMalloc(&h_z, M));
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
        h_y[n] = 2.34;
    }

    real *d_x, *d_y, *d_z;
    CHECK(hipMalloc(&d_x, M));
    CHECK(hipMalloc(&d_y, M));
    CHECK(hipMalloc(&d_z, M));

    for (int i = 0; i < MAX_NUM_STREAMS; i++)
    {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    for (int num = 1; num <= MAX_NUM_STREAMS; num *= 2)
    {
        timing(h_x, h_y, h_z, d_x, d_y, d_z, num);
    }

    for (int i = 0 ; i < MAX_NUM_STREAMS; i++)
    {
        CHECK(hipStreamDestroy(streams[i]));
    }

    CHECK(hipHostFree(h_x));
    CHECK(hipHostFree(h_y));
    CHECK(hipHostFree(h_z));
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));
    
    return 0;
}

void __global__ add(const real *x, const real *y, real *z, int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        for (int i = 0; i < 40; ++i)
        {
            z[n] = x[n] + y[n];
        }
    }
}

void timing(const real *h_x, const real *h_y, real *h_z,
            real *d_x, real *d_y, real *d_z, 
            const int num)
{
    int N1 = N / num;
    int M1 = M / num;
    
    float t_sum = 0;
    float t2_sum = 0;

    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        for (int i = 0; i < num; i++)
        {
            int offset = i * N1;
            CHECK(hipMemcpyAsync(d_x + offset, h_x + offset, M1, hipMemcpyHostToDevice, streams[i]));
            CHECK(hipMemcpyAsync(d_y + offset, h_y + offset, M1, hipMemcpyHostToDevice, streams[i]));
            
            int block_size = 128;
            int grid_size = (N1 - 1) / block_size + 1;
            add<<<grid_size, block_size, 0, streams[i]>>>
            (d_x + offset, d_y + offset, d_z + offset, N1);

            CHECK(hipMemcpyAsync(h_z + offset, d_z + offset, M1, hipMemcpyDeviceToHost, streams[i]));
        }

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));

        const float t_ave = t_sum / NUM_REPEATS;
        const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
        printf("%d %g\n", num, t_ave);
        
    }
}