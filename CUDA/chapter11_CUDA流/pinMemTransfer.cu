#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using CUDA's memory copy API to transfer data to and from the
 * device. In this case, hipMalloc is used to allocate memory on the GPU and
 * hipMemcpy is used to transfer the contents of host memory to an array
 * allocated using hipMalloc. Host memory is allocated using hipHostMalloc to
 * create a page-locked host array.
 */

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));

    // memory size
    unsigned int isize = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));

    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB canMap %d\n", dev,
           deviceProp.name, isize, nbytes / (1024.0f * 1024.0f),
           deviceProp.canMapHostMemory);

    // allocate pinned host memory
    float *h_a;
    CHECK(hipHostMalloc ((float **)&h_a, nbytes));

    // allocate device memory
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    // initialize host memory
    memset(h_a, 0, nbytes);

    for (int i = 0; i < isize; i++) h_a[i] = 100.10f;

    // transfer data from the host to the device
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));
    CHECK(hipHostFree(h_a));

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}