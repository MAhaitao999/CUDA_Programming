#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

/*
 * This example demonstrates overlapping computation and communication by
 * partitioning a data set and asynchronously launching the memory copies and
 * kernels for each subset. Launching all transfers and kernels for a given
 * subset in the same CUDA stream ensures that computation on the device is not
 * started until the necessary data has been transferred. However, because the
 * work of each subset is independent of all other subsets, the communication
 * and computation of different subsets will overlap.
 *
 * This example launches copies and kernels in depth-first order.
 */


#define NSTREAM 4
#define BDIM 128

void initialData(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {
        for (int i = 0; i < N; ++i)
        {
            C[idx] = A[idx] + B[idx];
        }
    }
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");
}

int main(int argc, char **argv)
{
    printf("> %s Starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
    {
        if (deviceProp.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                    "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // set up max connectioin
    char iname[] = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv (iname, "1", 1);
    char *ivalue =  getenv (iname);
    printf ("> %s = %s\n", iname, ivalue);
    printf ("> with streams = %d\n", NSTREAM);

    // set up data size of vectors
    int nElem = 1 << 18;
    printf("> vector size = %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // malloc pinned host memory for async memcpy
    float *h_A, *h_B, *hostRef, *gpuRef;
    CHECK(hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&hostRef, nBytes, hipHostMallocDefault));

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // invoke kernel at host side
    dim3 block (BDIM);
    dim3 grid  ((nElem + block.x - 1) / block.x);
    printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x,
            block.y);

    // sequential operation
    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float memcpy_h2d_time;
    CHECK(hipEventElapsedTime(&memcpy_h2d_time, start, stop));

    CHECK(hipEventRecord(start, 0));
    sumArrays<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float kernel_time;
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float memcpy_d2h_time;
    CHECK(hipEventElapsedTime(&memcpy_d2h_time, start, stop));
    float itotal = kernel_time + memcpy_h2d_time + memcpy_d2h_time;

    printf("\n");
    printf("Measured timings (throughput):\n");
    printf(" Memcpy host to device\t: %f ms (%f GB/s)\n",
           memcpy_h2d_time, (nBytes * 1e-6) / memcpy_h2d_time);
    printf(" Memcpy device to host\t: %f ms (%f GB/s)\n",
           memcpy_d2h_time, (nBytes * 1e-6) / memcpy_d2h_time);
    printf(" Kernel\t\t\t: %f ms (%f GB/s)\n",
           kernel_time, (nBytes * 2e-6) / kernel_time);
    printf(" Total\t\t\t: %f ms (%f GB/s)\n",
           itotal, (nBytes * 2e-6) / itotal);

    // grid parallel operation
    int iElem = nElem / NSTREAM;
    size_t iBytes = iElem * sizeof(float);
    grid.x = (iElem + block.x - 1) / block.x;

    hipStream_t stream[NSTREAM];

    for (int i = 0; i < NSTREAM; ++i)
    {
        CHECK(hipStreamCreate(&stream[i]));
    }

    CHECK(hipEventRecord(start, 0));

    // initiate all work on the device asynchronously in depth-first order
    for (int i = 0; i < NSTREAM; ++i)
    {
        int ioffset = i * iElem;
        CHECK(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes,
                              hipMemcpyHostToDevice, stream[i]));
        CHECK(hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes,
                              hipMemcpyHostToDevice, stream[i]));
        sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset],
                &d_C[ioffset], iElem);
        CHECK(hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes,
                              hipMemcpyDeviceToHost, stream[i]));
    }

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float execution_time;
    CHECK(hipEventElapsedTime(&execution_time, start, stop));

    printf("\n");
    printf("Actual results from overlapped data transfers:\n");
    printf(" overlap with %d streams : %f ms (%f GB/s)\n", NSTREAM,
           execution_time, (nBytes * 2e-6) / execution_time );
    printf(" speedup                : %f \n",
           ((itotal - execution_time) * 100.0f) / itotal);

    // check kernel error
    CHECK(hipGetLastError());

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    CHECK(hipHostFree(h_A));
    CHECK(hipHostFree(h_B));
    CHECK(hipHostFree(hostRef));
    CHECK(hipHostFree(gpuRef));

    // destroy events
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    // destroy streams
    for (int i = 0; i < NSTREAM; ++i)
    {
        CHECK(hipStreamDestroy(stream[i]));
    }

    CHECK(hipDeviceReset());
    return(0);
}