#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cmath>
#include <cstdio>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 10;
const int N1 = 1024;
const int MAX_NUM_STREAMS = 30;
const int N = N1 * MAX_NUM_STREAMS;
const int M = sizeof(real) * N;
const int block_size = 128;
const int grid_size = (N1 - 1) / block_size + 1;
hipStream_t streams[MAX_NUM_STREAMS];

void timing(const real *d_x, const real *d_y, real *d_z, const int num);

int main(int argc, char *argv[])
{

    real *h_x = (real *)malloc(M);
    real *h_y = (real *)malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
        h_y[n] = 2.34;
    }

    real *d_x, *d_y, *d_z;
    CHECK(hipMalloc(&d_x, M));
    CHECK(hipMalloc(&d_y, M));
    CHECK(hipMalloc(&d_z, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

    for (int n = 0; n < MAX_NUM_STREAMS; ++n)
    {
        CHECK(hipStreamCreate(&streams[n]));
    }

    for (int num = 1; num <= MAX_NUM_STREAMS; ++num)
    {
        timing(d_x, d_y, d_z, num);
    }

    for (int n = 0 ; n < MAX_NUM_STREAMS; ++n)
    {
        CHECK(hipStreamDestroy(streams[n]));
    }

    free(h_x);
    free(h_y);
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));
    
    return 0;
}

void __global__ add(const real *d_x, const real *d_y, real *d_z)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N1)
    {
        for (int i = 0; i < 100000; ++i)
        {
            d_z[n] = d_x[n] + d_y[n];
        }
    }
}

void timing(const real *d_x, const real *d_y, real *d_z, const int num)
{
    float t_sum = 0;
    float t2_sum = 0;

    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        for (int n = 0; n < num; ++n)
        {
            int offset = n * N1;
            add<<<grid_size, block_size, 0, streams[n]>>>
            (d_x + offset, d_y + offset, d_z + offset);
        }
 
        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("%g\n", t_ave);
}