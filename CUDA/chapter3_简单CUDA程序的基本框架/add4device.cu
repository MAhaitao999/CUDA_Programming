#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>


const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;
void __global__ add1(const double *x, const double *y, double *z, const int N);
void __global__ add2(const double *x, const double *y, double *z, const int N);
void __global__ add3(const double *x, const double *y, double *z, const int N);
void check(const double *z, int N);

int main(int argc, char *argv[])
{
    const int N = 100000001;
    const int M = sizeof(double) * N;
    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    add1<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    add2<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    add3<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    return 0;
}

// 版本一：有返回值的设备函数
double __device__ add1_device(const double x, const double y)
{
    return (x + y);
}

void __global__ add1(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = add1_device(x[n], y[n]);
    }
}

// 版本二：用指针的设备函数
void __device__ add2_device(const double x, const double y, double *z)
{
    *z = x + y;
}

void __global__ add2(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        add2_device(x[n], y[n], &z[n]);
    }
}

// 版本三：用引用（reference）的设备函数
void __device__ add3_device(const double x, const double y, double &z)
{
    z = x + y;
}

void __global__ add3(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        add3_device(x[n], y[n], z[n]);
    }
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

