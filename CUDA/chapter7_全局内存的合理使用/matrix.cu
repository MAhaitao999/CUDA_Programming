#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 10;
const int TILE_DIM = 32;

void timing(const real *d_A, real *d_B, const int N, const int task);

__global__ void copy(const real *A, real *B, const int N);
__global__ void transpose1(const real *A, real *B, const int N);
__global__ void transpose2(const real *A, real *B, const int N);
__global__ void transpose3(const real *A, real *B, const int N);
void print_matrix(const int N, const real *A);

int main(int argc, char *argv[])
{

    if (argc != 2)
    {
        printf("usage: %s N\n", argv[0]);
        exit(1);
    }
    const int N = atoi(argv[1]);

    const int N2 = N * N;
    const int M = sizeof(real) * N2;
    real *h_A = (real *) malloc(M);
    real *h_B = (real *) malloc(M);
    for (int n = 0; n < N2; ++n)
    {
        h_A[n] = n;
    }
    real *d_A, *d_B;
    CHECK(hipMalloc(&d_A, M));
    CHECK(hipMalloc(&d_B, M));
    CHECK(hipMemcpy(d_A, h_A, M, hipMemcpyHostToDevice));

    printf("\ncopy:\n");
    timing(d_A, d_B, N, 0);
    printf("\ntranspose with coalesced read:\n");
    timing(d_A, d_B, N, 1);
    printf("\ntranspose with coalesced write:\n");
    timing(d_A, d_B, N, 2);
    printf("\ntranspose with coalesced write and __ldg read:\n");
    timing(d_A, d_B, N, 3);

    CHECK(hipMemcpy(h_B, d_B, M, hipMemcpyDeviceToHost));
    if (N <= 10)
    {
        printf("A =\n");
        print_matrix(N, h_A);
        printf("\nB =\n");
        print_matrix(N, h_B);
    }

    free(h_A);
    free(h_B);
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    return 0;
}

__global__ void copy(const real *A, real *B, const int N)
{
    const int nx = blockIdx.x * TILE_DIM + threadIdx.x;
    const int ny = blockIdx.y * TILE_DIM + threadIdx.y;
    if (nx < N && ny < N)
    {
        B[nx * N + ny] = A[nx * N + ny];
    }    
}

__global__ void transpose1(const real *A, real *B, const int N)
{
    const int nx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ny = blockIdx.y * blockDim.y + threadIdx.y;
    if (nx < N && ny < N)
    {
        B[nx * N + ny] = A[ny * N + nx];
    }
}

__global__ void transpose2(const real *A, real *B, const int N)
{
    const int nx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ny = blockIdx.y * blockDim.y + threadIdx.y;
    if (nx < N && ny < N)
    {
        B[ny * N + nx] = A[nx * N + ny];
    }
}

__global__ void transpose3(const real *A, real *B, const int N)
{
    const int nx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ny = blockIdx.y * blockDim.y + threadIdx.y;
    if (nx < N && ny < N)
    {
        B[ny * N + nx] = __ldg(&A[nx * N + ny]);
    }
}

void timing(const real *d_A, real *d_B, const int N, const int task)
{
    const int grid_size_x = (N + TILE_DIM - 1) / TILE_DIM;
    const int grid_size_y = grid_size_x;
    const dim3 block_size(TILE_DIM, TILE_DIM);
    const dim3 grid_size(grid_size_x, grid_size_y);

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        switch (task)
        {
            case 0:
                copy<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 1:
                transpose1<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 2:
                transpose2<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 3:
                transpose3<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            default:
                printf("Error: wrong task\n");
                exit(1);
                break;
        }

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);
}

void print_matrix(const int N, const real *A)
{
    for (int ny = 0; ny < N; ny++)
    {
        for (int nx = 0; nx < N; nx++)
        {
            printf("%g\t", A[ny * N + nx]);
        }
        printf("\n");
    }
}
