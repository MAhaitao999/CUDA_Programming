#include "hip/hip_runtime.h"
#include "error.cuh"
#include "force.h"
#include "mic.h"

struct LJ
{
    real cutoff2;
    real e24s6;
    real e48s12;
    real e4s6;
    real e4s12;
};

static void __global__ gpu_find_force
(
    LJ lj, int N, int *g_NN, int *g_NL, Box box,
    real *g_x, real *g_y, real *g_z,
    real *g_fx, real *g_fy, real *g_fz, real *g_pe
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        real fx = 0.0;
        real fy = 0.0;
        real fz = 0.0;
        real potential = 0.0;
        int NN = g_NN[i];
        real x_i = g_x[i];
        real y_i = g_y[i];
        real z_i = g_z[i];
        for (int k = 0; k < NN; ++k)
        {
            int j = g_NL[i + N * k];
            real x_ij  = g_x[j] - x_i;
            real y_ij  = g_y[j] - y_i;
            real z_ij  = g_z[j] - z_i;
            apply_mic(box, &x_ij, &y_ij, &z_ij);
            real r2 = x_ij*x_ij + y_ij*y_ij + z_ij*z_ij;
            if (r2 > lj.cutoff2) { continue; }

            real r2inv = 1.0 / r2;
            real r4inv = r2inv * r2inv;
            real r6inv = r2inv * r4inv;
            real r8inv = r4inv * r4inv;
            real r12inv = r4inv * r8inv;
            real r14inv = r6inv * r8inv;
            real f_ij = lj.e24s6 * r8inv - lj.e48s12 * r14inv;
            potential += lj.e4s12 * r12inv - lj.e4s6 * r6inv;
            fx += f_ij * x_ij;
            fy += f_ij * y_ij;
            fz += f_ij * z_ij;
        }
        g_fx[i] = fx;
        g_fy[i] = fy;
        g_fz[i] = fz;
        g_pe[i] = potential * 0.5;
    }
}

void find_force(int N, int MN, Atom *atom)
{
    const real epsilon = 1.032e-2;
    const real sigma = 3.405;
    const real cutoff = 10.0;
    const real cutoff2 = cutoff * cutoff;
    const real sigma_3 = sigma * sigma * sigma;
    const real sigma_6 = sigma_3 * sigma_3;
    const real sigma_12 = sigma_6 * sigma_6;
    const real e24s6 = 24.0 * epsilon * sigma_6;
    const real e48s12 = 48.0 * epsilon * sigma_12;
    const real e4s6 = 4.0 * epsilon * sigma_6;
    const real e4s12 = 4.0 * epsilon * sigma_12;
    LJ lj;
    lj.cutoff2 = cutoff2;
    lj.e24s6 = e24s6;
    lj.e48s12 = e48s12;
    lj.e4s6 = e4s6;
    lj.e4s12 = e4s12;

    Box box;
    box.lx = atom->box[0];
    box.ly = atom->box[1];
    box.lz = atom->box[2];
    box.lx2 = atom->box[3];
    box.ly2 = atom->box[4];
    box.lz2 = atom->box[5];

    int m = sizeof(real) * N;
    CHECK(hipMemcpy(atom->g_x, atom->x, m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_y, atom->y, m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_z, atom->z, m, hipMemcpyHostToDevice));

    int block_size = 128;
    int grid_size = (N - 1) / block_size + 1;
    gpu_find_force<<<grid_size, block_size>>>
    (
        lj, N,  atom->g_NN, atom->g_NL, box,
        atom->g_x, atom->g_y, atom->g_z,
        atom->g_fx, atom->g_fy, atom->g_fz, atom->g_pe
    );

    CHECK(hipMemcpy(atom->fx, atom->g_fx, m, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(atom->fy, atom->g_fy, m, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(atom->fz, atom->g_fz, m, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(atom->pe, atom->g_pe, m, hipMemcpyDeviceToHost));
}


