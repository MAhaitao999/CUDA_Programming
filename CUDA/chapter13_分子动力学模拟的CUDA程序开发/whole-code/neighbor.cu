#include "hip/hip_runtime.h"
#include "neighbor.h"
#include "mic.h"
#include <stdio.h>
#include <stdlib.h>

static void __global__ gpu_find_neighbor
(
    int N, int MN, int *g_NN, int *g_NL, Box box, 
    real *g_x, real *g_y, real *g_z, real cutoff2
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (n1 < N)
    {
        int count = 0;
        real x1 = g_x[n1];
        real y1 = g_y[n1];
        real z1 = g_z[n1];
        for (int n2 = 0; n2 < N; n2++)
        {
            real x12 = g_x[n2] - x1;
            real y12 = g_y[n2] - y1;
            real z12 = g_z[n2] - z1;
            apply_mic(box, &x12, &y12, &z12);
            real d12_square = x12*x12 + y12*y12 + z12*z12;
            if ((n2 != n1) && (d12_square < cutoff2))
            {
                g_NL[count++ * N + n1] = n2;
            }
        }
        g_NN[n1] = count;
    }
}

void find_neighbor(int N, int MN, Atom *atom)
{
    real cutoff = 11.0;
    real cutoff2 = cutoff * cutoff;

    Box box;
    box.lx = atom->box[0];
    box.ly = atom->box[1];
    box.lz = atom->box[2];
    box.lx2 = atom->box[3];
    box.ly2 = atom->box[4];
    box.lz2 = atom->box[5];

    int block_size = 128;
    int grid_size = (N - 1) / block_size + 1;
    gpu_find_neighbor<<<grid_size, block_size>>>
    (
        N, MN, atom->g_NN, atom->g_NL, box,
        atom->g_x, atom->g_y, atom->g_z, cutoff2
    );
}

