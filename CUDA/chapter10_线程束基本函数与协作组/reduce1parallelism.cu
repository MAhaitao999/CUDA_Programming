#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 100;
const int N = 100000000;
const int M = sizeof(real) * N;
const int BLOCK_SIZE = 128;
const int GRID_SIZE = 10240;

void timing(const real *h_x);

int main(void)
{
    real *h_x = (real *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
    }
    real *d_x;
    CHECK(hipMalloc(&d_x, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

    timing(d_x);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void __global__ reduce_cp(const real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    extern __shared__ real s_y[];

    real y = 0.0;
    const int stride = blockDim.x * gridDim.x;
    for (int n = bid * blockDim.x + tid; n < N; n += stride)
    {
        y += d_x[n];
    }
    s_y[tid] = y;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    y = s_y[tid];

    thread_block_tile<32> g = tiled_partition<32>(this_thread_block());
    for (int i = g.size() >> 1; i > 0; i >>= 1)
    {
        y += g.shfl_down(y, i);
    }

    if (tid == 0)
    {
        d_y[bid] = y;
    }
}

real reduce(const real *d_x)
{
    const int ymem = sizeof(real) * GRID_SIZE;
    const int smem = sizeof(real) * BLOCK_SIZE;

    real h_y[1] = {0};
    real *d_y;
    CHECK(hipMalloc(&d_y, ymem));

    reduce_cp<<<GRID_SIZE, BLOCK_SIZE, smem>>>(d_x, d_y, N);
    reduce_cp<<<1, 1024, sizeof(real) * 1024>>>(d_y, d_y, GRID_SIZE);

    CHECK(hipMemcpy(h_y, d_y, sizeof(real), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_y));

    return h_y[0];
}

void timing(const real *d_x)
{
    real sum = 0;

    for (int repeat = 0; repeat < NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        sum = reduce(d_x); 

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    printf("sum = %f.\n", sum);
}