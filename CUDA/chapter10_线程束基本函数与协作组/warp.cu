#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cstdio>

const unsigned WIDTH = 8;
const unsigned BLOCK_SIZE = 16;
const unsigned FULL_MASK = 0xffffffff;

void __global__ test_warp_primitives(void);

int main(int argc, char *argv[])
{

    test_warp_primitives<<<1, BLOCK_SIZE>>>();
    CHECK(hipDeviceSynchronize());

    return 0;
}

void __global__ test_warp_primitives(void)
{
    int tid = threadIdx.x;
    int lane_id = tid % WIDTH;

    if (tid == 0) printf("threadIdx.x: ");
    printf("%2d ", tid);
    if (tid == 0) printf("\n");

    if (tid == 0) printf("lane_id:     ");
    printf("%2d ", lane_id);
    if (tid == 0) printf("\n");

    unsigned mask1 = __ballot_sync(FULL_MASK, tid > 0);
    unsigned mask2 = __ballot_sync(FULL_MASK, tid == 0);
    if (tid == 0) printf("FULL_MASK = %x\n", FULL_MASK);
    if (tid == 1) printf("mask1     = %x\n", mask1);
    if (tid == 0) printf("mask2     = %x\n", mask2);

    int result = __all_sync(FULL_MASK, tid);
    if (tid == 0) printf("all_sync (FULL_MASK): %d\n", result);

    result = __all_sync(mask1, tid);
    if (tid == 1) printf("all_sync     (mask1): %d\n", result);

    result = __any_sync(FULL_MASK, tid);
    if (tid == 0) printf("any_sync (FULL_MASK): %d\n", result);

    result = __any_sync(mask2, tid);
    if (tid == 0) printf("any_sync     (mask2): %d\n", result);

    int value = __shfl_sync(FULL_MASK, tid, 2, WIDTH);
    if (tid == 0) printf("shfl:      ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_up_sync(FULL_MASK, tid, 1, WIDTH);
    if (tid == 0) printf("shfl_up:   ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_down_sync(FULL_MASK, tid, 1, WIDTH);
    if (tid == 0) printf("shfl_down: ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_xor_sync(FULL_MASK, tid, 1, WIDTH);
    if (tid == 0) printf("shfl_xor:  ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");
}