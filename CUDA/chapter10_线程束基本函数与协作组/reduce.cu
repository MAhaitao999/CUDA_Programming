#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cstdio>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 100;
const int N = 100000000;
const int M = sizeof(real) * N;
const int BLOCK_SIZE = 128;
const unsigned FULL_MASK = 0xffffffff;

void timing(const real *d_x, const int method);

int main(int argc, char *argv[])
{
    real *h_x = (real *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
    }
    real *d_x;
    CHECK(hipMalloc(&d_x, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

    printf("\nusing syncwarp:\n");
    timing(d_x, 0);
    printf("\nusing shfl:\n");
    timing(d_x, 1);
    printf("\nusing cooperative group:\n");
    timing(d_x, 2);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void __global__ reduce_syncwarp(const real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    extern __shared__ real s_y[];
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads(); // 一个线程块内的所有线程同步
    } // 这一步执行完留下0-31号线程

    for (int offset = 16; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncwarp();
    }

    if (tid == 0)
    {
        atomicAdd(d_y, s_y[0]);
    }

}

void __global__ reduce_shfl(const real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    extern __shared__ real s_y[];
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    } // 这一步执行完留下0-31号线程

    real y = s_y[tid];

    for (int offset = 16; offset > 0; offset >>= 1)
    {
        y += __shfl_down_sync(FULL_MASK, y, offset);
    }

    if (tid == 0)
    {
        atomicAdd(d_y, y);
    }

}

void __global__ reduce_cp(const real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    extern __shared__ real s_y[];
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    real y = s_y[tid];

    thread_block_tile<32> g = tiled_partition<32>(this_thread_block());
    for (int i = g.size() >> 1; i > 0; i>>= 1)
    {
        y += g.shfl_down(y, i);
    }

    if (tid == 0)
    {
        atomicAdd(d_y, y);
    }
}

real reduce(const real *d_x, const int method)
{
    const int grid_size = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const int smem = sizeof(real) * BLOCK_SIZE;

    real h_y[1] = {0};
    real *d_y;
    CHECK(hipMalloc(&d_y, sizeof(real)));
    CHECK(hipMemcpy(d_y, h_y, sizeof(real), hipMemcpyHostToDevice));

    switch (method)
    {
        case 0:
            reduce_syncwarp<<<grid_size, BLOCK_SIZE, smem>>>(d_x, d_y, N);
            break;
        case 1:
            reduce_shfl<<<grid_size, BLOCK_SIZE, smem>>>(d_x, d_y, N);
            break;
        case 2:
            reduce_cp<<<grid_size, BLOCK_SIZE, smem>>>(d_x, d_y, N);
            break;
        default:
            printf("Wrong method.\n");
            exit(1);
    }

    CHECK(hipMemcpy(h_y, d_y, sizeof(real), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_y));

    return h_y[0];
}

void timing(const real *d_x, const int method)
{
    real sum = 0;
    
    for (int repeat = 0; repeat < NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        sum = reduce(d_x, method); 

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    printf("sum = %f.\n", sum);
}


