
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <hiprand/hiprand.h>

void output_results(int N, double *g_x);

int main(int argc, char *argv[])
{
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, 1234);
    int N = 100000;
    double *g_x;
    hipMalloc((void **)&g_x, sizeof(double) * N);
    hiprandGenerateNormalDouble(generator, g_x, N, 0.0, 1.0);
    double *x = (double *)calloc(N, sizeof(double));
    hipMemcpy(x, g_x, sizeof(double) * N, hipMemcpyDeviceToHost);
    hipFree(g_x);
    output_results(N, x);
    free(x);

    return 0;
}

void output_results(int N, double *x)
{
    FILE *fid = fopen("x2.txt", "w");
    for(int n = 0; n < N; n++)
    {
        fprintf(fid, "%g\n", x[n]);
    }
    fclose(fid);
}