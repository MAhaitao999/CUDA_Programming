#include "error.cuh"
#include <cstdio>
#include <cstdlib>
#include <hipsolver.h>

int main(int argc, char *argv[])
{
    int N = 2;
    int N2 = N * N;
    hipDoubleComplex *A_cpu = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex) * N2);

    for (int n = 0; n < N2; ++n)
    {
        A_cpu[0].x = 0;
        A_cpu[1].x = 0;
        A_cpu[2].x = 0;
        A_cpu[3].x = 0;
        A_cpu[0].y = 0;
        A_cpu[1].y = 1;
        A_cpu[2].y = -1;
        A_cpu[3].y = 0;
    }

    hipDoubleComplex *A;
    CHECK(hipMalloc((void**)&A, sizeof(hipDoubleComplex) * N2));
    CHECK(hipMemcpy(A, A_cpu, sizeof(hipDoubleComplex) * N2, hipMemcpyHostToDevice));

    double *W_cpu = (double*)malloc(sizeof(double) * N);
    double *W;
    CHECK(hipMalloc((void**)&W, sizeof(double) * N));

    hipsolverHandle_t handle = NULL;
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    int lwork = 0;
    hipsolverDnZheevd_bufferSize(handle, jobz, uplo, N, A, N, W, &lwork);
    hipDoubleComplex* work;
    CHECK(hipMalloc((void **)&work, sizeof(hipDoubleComplex) * lwork));

    int *info;
    CHECK(hipMalloc((void **)&info, sizeof(int)));
    hipsolverDnZheevd(handle, jobz, uplo, N, A, N, W, work, lwork, info);
    hipMemcpy(W_cpu, W, sizeof(double) * N, hipMemcpyDeviceToHost);

    printf("Eigenvalues are: \n");
    for (int n = 0; n < N; ++n)
    {
        printf("%g\n", W_cpu[n]);
    }

    hipsolverDnDestroy(handle);

    free(A_cpu);
    free(W_cpu);
    CHECK(hipFree(A));
    CHECK(hipFree(W));
    CHECK(hipFree(work));
    CHECK(hipFree(info));

    return 0;
}