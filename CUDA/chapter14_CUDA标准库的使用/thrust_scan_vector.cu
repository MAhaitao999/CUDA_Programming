#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <cstdio>

int main(void)
{
    int N = 10;
    thrust::device_vector<int> x(N, 0);
    thrust::device_vector<int> y(N, 0);
    for (int i = 0; i < x.size(); ++i)
    {
        x[i] = i + 1;
    }
    thrust::inclusive_scan(x.begin(), x.end(), y.begin());
    for (int i = 0; i < y.size(); ++i)
    {
        printf("%d ", (int) y[i]);
    }
    printf("\n");
    return 0;
}