#include "error.cuh" 
#include <stdio.h>
#include <hipblas.h>

void print_matrix(int R, int C, double* A, const char* name);

int main(void)
{
    int M = 2;
    int K = 3;
    int N = 2;
    int MK = M * K;
    int KN = K * N;
    int MN = M * N;

    double *h_A = (double*) malloc(sizeof(double) * MK);
    double *h_B = (double*) malloc(sizeof(double) * KN);
    double *h_C = (double*) malloc(sizeof(double) * MN);
    for (int i = 0; i < MK; i++)
    {
        h_A[i] = i;
    }
    print_matrix(M, K, h_A, "A");
    for (int i = 0; i < KN; i++)
    {
        h_B[i] = i;
    }
    print_matrix(K, N, h_B, "B");
    for (int i = 0; i < MN; i++)
    {
        h_C[i] = 0;
    }

    double *g_A, *g_B, *g_C;
    CHECK(hipMalloc((void **)&g_A, sizeof(double) * MK));
    CHECK(hipMalloc((void **)&g_B, sizeof(double) * KN));
    CHECK(hipMalloc((void **)&g_C, sizeof(double) * MN));

    hipblasSetVector(MK, sizeof(double), h_A, 1, g_A, 1);
    hipblasSetVector(KN, sizeof(double), h_B, 1, g_B, 1);
    hipblasSetVector(MN, sizeof(double), h_C, 1, g_C, 1);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double alpha = 1.0;
    double beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        M, N, K, &alpha, g_A, M, g_B, K, &beta, g_C, M);
    hipblasDestroy(handle);

    hipblasGetVector(MN, sizeof(double), g_C, 1, h_C, 1);
    print_matrix(M, N, h_C, "C = A x B");

    free(h_A);
    free(h_B);
    free(h_C);
    CHECK(hipFree(g_A));
    CHECK(hipFree(g_B));
    CHECK(hipFree(g_C));
    return 0;
}

void print_matrix(int R, int C, double* A, const char* name)
{
    printf("%s = \n", name);
    for (int r = 0; r < R; ++r)
    {
        for (int c = 0; c < C; ++c)
        {
            printf("%10.6f", A[c * R + r]);
        }
        printf("\n");
    }
}