#include <thrust/execution_policy.h>
#include <thrust/scan.h>
#include <cstdio>

int main(int argc, char *argv[])
{
    int N = 10;
    int *x, *y;
    hipMalloc((void **)&x, sizeof(int) * N);
    hipMalloc((void **)&y, sizeof(int) * N);
    int *h_x = (int *)malloc(sizeof(int) * N);
    for (int i = 0; i < N; ++i)
    {
        h_x[i] = i + 1;
    }
    hipMemcpy(x, h_x, sizeof(int) * N, hipMemcpyHostToDevice);

    thrust::inclusive_scan(thrust::device, x, x + N, y);

    int *h_y = (int *)malloc(sizeof(int) * N);
    hipMemcpy(h_y, y, sizeof(int) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
        printf("%d ", h_y[i]);
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    free(h_x);
    free(h_y);
    return 0;
}